#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define inputHeight 5  // 输入高度
#define inputWidth 5   // 输入宽度
#define inputChannels 3 // 输入通道数
#define kernelSize 3   // 卷积核大小
#define kernelNum 3    // 卷积核数量
#define stride 1       // 步长
#define TILESIZE 16    // 线程块大小

// 2D卷积核函数
__global__ void conv2d_global(float* im2col, float* kernel, float* output) {
    // 计算输出的高度和宽度
    int outHeight = (inputHeight - kernelSize) / stride + 1;
    int outWidth = (inputWidth - kernelSize) / stride + 1;

    // 计算线程的全局索引
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // 如果线程的索引在输出范围内，执行卷积操作
    if (row < outHeight && col < outWidth) {
        //printf("%d %d\n",row,col);
        for (int kn = 0; kn < kernelNum; ++kn) {
            float sum = 0.0f; // 初始化卷积和为0
            // 对每个输入通道和卷积核进行迭代
            int n = 0;
            for (int kc = 0; kc < inputChannels; ++kc) {
                
                for (int i = 0; i < kernelSize; ++i) {
                    for (int j = 0; j < kernelSize; ++j) {
                        int length = inputChannels * kernelSize * kernelSize;

                        // 计算卷积和
                        sum += im2col[(row * outWidth + col) * length + n] * 
                                kernel[((kn * inputChannels + kc) * kernelSize + i) * kernelSize + j];
                        n++;
 
                    }
                }
               
            }
            // 将计算结果存储到输出数组
            output[(kn * outHeight + row) * outWidth + col] = sum;
        }
    }
}

// 输出input
void printInput(float* input){
    printf("The input:\n\n");
    for(int i = 0; i < inputChannels ; i++){
        printf("The %dth channel of input:\n",i + 1);
        for(int j = 0; j < inputHeight ; j++){
            for(int k = 0 ; k < inputWidth ;k++ ){
                printf("%lf ",input[i * inputHeight * inputWidth + j * inputWidth + k]);
            }
            printf("\n");
        }
    }
    printf("\n");
}


// 输出kernel
void printKernel(float* kernel){
    printf("The kernel:\n\n");
    for(int d = 0 ;d < kernelNum; d++){
        printf("The %dth kernel:\n",d + 1);
        for(int i = 0; i < inputChannels ; i++){
            printf("The %dth channel of the kernel:\n",i + 1);
            for(int j = 0; j < kernelSize ; j++){
                for(int k = 0 ; k < kernelSize ;k++ ){
                    printf("%lf ",kernel[i * kernelSize * kernelSize + j * kernelSize + k]);
                }
                printf("\n");
            }
        }
        printf("\n");
    }    
    printf("\n");
}

// 输出output
void printOutput(float* output,int outputHeight,int outputWidth){
    printf("The output:\n\n");
    for (int kn = 0; kn < kernelNum; ++kn) {
        printf("Channel %d of the output:\n", kn + 1);
        for (int i = 0; i < outputHeight; ++i) {
            for (int j = 0; j < outputWidth; ++j) {
                printf("%f ", output[(kn * outputHeight + i) * outputWidth + j]);
            }
            printf("\n");
        }
        printf("\n");
    }
    printf("\n");
}

// 输出im2col
void printIm2col(float* im2col,int outputHeight,int outputWidth){
    printf("The im2col:\n\n");
    for (int kn = 0; kn < outputHeight * outputWidth ; ++kn) {
        for (int i = 0; i < kernelSize * kernelSize * inputChannels ; ++i) {
            printf("%f ",im2col[kn * kernelSize * kernelSize * inputChannels + i]);
        }
        printf("\n");
    }
    printf("\n");
}

// 打印卷积层参数的函数
void printConvolutionParameters() {
    int outputHeight = (inputHeight - kernelSize) / stride + 1;
    int outputWidth = (inputWidth - kernelSize) / stride + 1;
    printf("Convolution Layer Parameters:\n");
    printf("Input Channels: %d\n", inputChannels);
    printf("Input Dimensions: %dx%d\n", inputHeight, inputWidth);
    printf("Kernel Size: %d\n", kernelSize);
    printf("Kernel Dimensions: %dx%d\n", kernelSize, kernelSize);
    printf("Kernel Number: %d\n", kernelNum);
    printf("Output Channels: %d\n", kernelNum);
    printf("Output Dimensions: %dx%d\n", outputHeight, outputWidth);
    printf("Stride: %d\n", stride);
    printf("Thread Block Size: %dx%d\n", TILESIZE, TILESIZE);
    printf("Grid Size: %dx%d\n\n", (outputWidth + TILESIZE - 1) / TILESIZE, (outputHeight + TILESIZE - 1) / TILESIZE);
}

float* im_to_col(float*input,int outputHeight,int outputWidth){
    size_t row = outputHeight * outputWidth ;
    size_t col = kernelSize * kernelSize * inputChannels; 
    float *im2col = (float*)malloc(row * col * sizeof(float));
    for(int i = 0; i < outputHeight ; i++ ){
        for(int j = 0 ; j < outputWidth ; j++){
            int n = 0 ;
            for(int c = 0 ;c < inputChannels ; c++){
                    for(int k1 = 0; k1 < kernelSize ;k1++){
                        for(int k2 = 0 ;k2 < kernelSize ;k2++){
                            int startRow = i * stride;
                            int startCol = j * stride;
                            int rindex = startRow + k1;
                            int cindex = startCol +k2;
                            im2col[ (i * outputWidth +j) * col  + n] =
                                input[c * inputHeight * inputWidth + rindex * inputWidth + cindex];
                                n++;
                        }
                    }
                }
            }
            
            }

    return im2col;
}


int main() {
    // 定义输入、卷积核和输出的大小
    size_t inputSize = inputHeight * inputWidth * inputChannels;
    size_t kernelSizeTotal = kernelNum * kernelSize * kernelSize * inputChannels;
    size_t outputHeight = (inputHeight - kernelSize) / stride + 1;
    size_t outputWidth = (inputWidth - kernelSize) / stride + 1;
    size_t outputSize = outputHeight * outputWidth * kernelNum;

    printConvolutionParameters();

    // 分配主机内存
    float *input = (float *)malloc(inputSize * sizeof(float));
    float *kernel = (float *)malloc(kernelSizeTotal * sizeof(float));
    float *output = (float *)malloc(outputSize * sizeof(float));

    // 使用随机值初始化输入和卷积核
    srand((unsigned int)time(NULL));
    int x = 0; 
    for (int i = 0; i < inputSize; ++i) {
        //input[i] = (float)(rand() % 1000) / 1000.0f;
        input[i] = 1.0f;
        //input[i] = x++;
    }
    for (int i = 0; i < kernelSizeTotal; ++i) {
        //kernel[i] = (float)(rand() % 1000) / 1000.0f;
        kernel[i] = 1.0f;
    }

    // 输出输入
    //printInput(input);
    // 输出卷积核
    //printKernel(kernel);
    

    float *im2col = im_to_col(input,outputHeight,outputWidth); 
    // 输出im2col
    // printIm2col(im2col,outputHeight,outputWidth);

    // 分配CUDA内存
    float *d_input, *d_kernel, *d_output;
    //cudaMalloc((void **)&d_input, inputSize * sizeof(float));
    hipMalloc((void **)&d_input, outputHeight * outputWidth * kernelSize * kernelSize * inputChannels * sizeof(float));
    hipMalloc((void **)&d_kernel, kernelSizeTotal * sizeof(float));
    hipMalloc((void **)&d_output, outputSize * sizeof(float));

    // 从主机复制数据到设备
    hipMemcpy(d_input, im2col, outputHeight * outputWidth * kernelSize * kernelSize * inputChannels * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, kernelSizeTotal * sizeof(float), hipMemcpyHostToDevice);

    // 设置线程块和网格大小
    dim3 threadsPerBlock(TILESIZE, TILESIZE);
    dim3 numBlocks((outputWidth + TILESIZE - 1) / TILESIZE, (outputHeight + TILESIZE - 1) / TILESIZE);

    // 创建 CUDA 事件计时器
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // 记录开始时间
    hipEventRecord(start);

    // 启动CUDA核函数
    // 全局内存
    conv2d_global<<<numBlocks, threadsPerBlock>>>(d_input, d_kernel, d_output);

    // 同步线程，等待内核函数执行完成
    hipDeviceSynchronize();

     // 记录结束时间
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // 计算执行时间
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);


    // 从设备复制结果回主机
    hipMemcpy(output, d_output, outputSize * sizeof(float), hipMemcpyDeviceToHost);

    // 输出结果
    printOutput(output,outputHeight,outputWidth);

    
    // 输出计算时间
    printf("\nTime for  convolution: %.2f ms\n", milliseconds);

    // 释放CUDA内存
    hipFree(d_input);
    hipFree(d_kernel);
    hipFree(d_output);

    // 释放主机内存
    free(input);
    free(kernel);
    free(output);
    free(im2col);

    return 0;
}