#include <iostream>
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <chrono>

// CUDA 错误检查
#define CHECK_CUDA(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl; \
            std::exit(EXIT_FAILURE); \
        } \
    } while (0)

// cuDNN 错误检查
#define CHECK_CUDNN(call) \
    do { \
        hipdnnStatus_t err = call; \
        if (err != HIPDNN_STATUS_SUCCESS) { \
            std::cerr << "cuDNN error: " << hipdnnGetErrorString(err) << std::endl; \
            std::exit(EXIT_FAILURE); \
        } \
    } while (0)

int main() {
    // 初始化 cuDNN
    hipdnnHandle_t cudnn;
    CHECK_CUDNN(hipdnnCreate(&cudnn));

    // 输入特征图参数
    const int batch_size = 1;
    const int channels = 3;
    const int height = 5;
    const int width = 5;

    // 卷积核参数
    const int kernel_size = 3;
    const int kernel_channels = 3;
    const int num_kernels = 2;

    // 初始化输入数据和卷积核为 1
    float input[batch_size * channels * height * width];
    float kernel[num_kernels * kernel_channels * kernel_size * kernel_size];
    std::fill_n(input, batch_size * channels * height * width, 1.0f);
    std::fill_n(kernel, num_kernels * kernel_channels * kernel_size * kernel_size, 1.0f);

    // 分配 GPU 内存
    float *d_input, *d_kernel, *d_output;
    CHECK_CUDA(hipMalloc(&d_input, sizeof(input)));
    CHECK_CUDA(hipMalloc(&d_kernel, sizeof(kernel)));

    // 输出特征图尺寸
    int output_height = height - kernel_size + 1;
    int output_width = width - kernel_size + 1;
    CHECK_CUDA(hipMalloc(&d_output, batch_size * num_kernels * output_height * output_width * sizeof(float)));

    // 将数据从主机复制到设备
    CHECK_CUDA(hipMemcpy(d_input, input, sizeof(input), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_kernel, kernel, sizeof(kernel), hipMemcpyHostToDevice));

    // 创建输入特征图描述符
    hipdnnTensorDescriptor_t input_desc;
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&input_desc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, channels, height, width));

    // 创建卷积核描述符
    hipdnnFilterDescriptor_t kernel_desc;
    CHECK_CUDNN(hipdnnCreateFilterDescriptor(&kernel_desc));
    CHECK_CUDNN(hipdnnSetFilter4dDescriptor(kernel_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, num_kernels, kernel_channels, kernel_size, kernel_size));

    // 创建卷积描述符
    hipdnnConvolutionDescriptor_t conv_desc;
    CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CHECK_CUDNN(hipdnnSetConvolution2dDescriptor(conv_desc, 0, 0, 1, 1, 1, 1, HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT));

    // 创建输出特征图描述符
    hipdnnTensorDescriptor_t output_desc;
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&output_desc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, num_kernels, output_height, output_width));

    // 卷积算法选择
    hipdnnConvolutionFwdAlgo_t conv_algo;
    CHECK_CUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn, input_desc, kernel_desc, conv_desc, output_desc, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &conv_algo));

    // 分配工作空间
    size_t workspace_size;
    CHECK_CUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn, input_desc, kernel_desc, conv_desc, output_desc, conv_algo, &workspace_size));

    void *d_workspace;
    CHECK_CUDA(hipMalloc(&d_workspace, workspace_size));

    // 记录开始时间
    auto start = std::chrono::high_resolution_clock::now();

    // 执行卷积
    const float alpha = 1.0f, beta = 0.0f;
    CHECK_CUDNN(hipdnnConvolutionForward(cudnn, &alpha, input_desc, d_input, kernel_desc, d_kernel, conv_desc, conv_algo, d_workspace, workspace_size, &beta, output_desc, d_output));

    // 记录结束时间
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> duration_ms = end - start;

    // 将结果从设备复制到主机
    float output[batch_size * num_kernels * output_height * output_width];
    CHECK_CUDA(hipMemcpy(output, d_output, sizeof(output), hipMemcpyDeviceToHost));

    // 打印输出结果
    std::cout << "Output:\n";
    for (int i = 0; i < batch_size * num_kernels * output_height * output_width; ++i) {
        std::cout << output[i] << " ";
        if ((i + 1) % output_width == 0) std::cout << "\n";
        if ((i + 1) % (output_height * output_width) == 0) std::cout << "\n";
    }

    // 打印卷积时间
    std::cout << "Convolution time: " << duration_ms.count() << " ms\n";

    // 清理资源
    CHECK_CUDA(hipFree(d_input));
    CHECK_CUDA(hipFree(d_kernel));
    CHECK_CUDA(hipFree(d_output));
    CHECK_CUDA(hipFree(d_workspace));
    CHECK_CUDNN(hipdnnDestroyTensorDescriptor(input_desc));
    CHECK_CUDNN(hipdnnDestroyFilterDescriptor(kernel_desc));
    CHECK_CUDNN(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CHECK_CUDNN(hipdnnDestroyTensorDescriptor(output_desc));
    CHECK_CUDNN(hipdnnDestroy(cudnn));

    return 0;
}
